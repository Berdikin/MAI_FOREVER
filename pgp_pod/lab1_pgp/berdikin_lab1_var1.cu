
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel(double *arr1, double *arr2, int n) {
	int i, idx = blockDim.x * blockIdx.x + threadIdx.x;			// Абсолютный номер потока
	int offset = blockDim.x * gridDim.x;						// Общее кол-во потоков
	for(i = idx; i < n; i += offset)	
		arr1[i] = arr1[i] < arr2[i] ? arr2[i] : arr1[i];
}

int main() {

	int i, n = 0;
	scanf("%d", &n);
	double *arr1 = (double *)malloc(sizeof(double) * n);
	double *arr2 = (double *)malloc(sizeof(double) * n);

	
	for(i = 0; i < n; i++){
		scanf("%lf", &arr1[i]);
	}

	for(i = 0; i < n; i++){
		scanf("%lf", &arr2[i]);
	}		

	double *dev_arr1, *dev_arr2;
	hipMalloc(&dev_arr1, sizeof(double) * n);
	hipMemcpy(dev_arr1, arr1, sizeof(double) * n, hipMemcpyHostToDevice);

	hipMalloc(&dev_arr2, sizeof(double) * n);
	hipMemcpy(dev_arr2, arr2, sizeof(double) * n, hipMemcpyHostToDevice);

	kernel<<<256, 256>>>(dev_arr1, dev_arr2, n);

	hipMemcpy(arr1, dev_arr1, sizeof(double) * n, hipMemcpyDeviceToHost);
	hipFree(dev_arr1);

	hipMemcpy(arr2, dev_arr2, sizeof(double) * n, hipMemcpyDeviceToHost);
	hipFree(dev_arr2);
	for(i = 0; i < n; i++)
		printf("%.10f ", arr1[i]);
	printf("\n");
	free(arr1);
	free(arr2);
	return 0;
}
